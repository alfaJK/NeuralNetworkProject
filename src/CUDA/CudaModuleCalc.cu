#include "hip/hip_runtime.h"
#include "CUDA/CudaModuleCalc.h"
#include "stdio.h"
#include <time.h> 
#include <iostream>

// calc conv module

__device__ float d_ConvDirectDist(int index_x, int index_y , CudaTensor *input , CudaTensor* filter){
    float sum = 0;
    int Padding = (filter->Size.width - 1) / 2;
    for (int i = 0; i < filter->Size.height; i++) {
        for (int j = 0; j < filter->Size.width; j++) {
            int i0 = index_x + i - Padding;
            int j0 = index_y + j - Padding;
            if (i0 < 0 || i0 >= input->Size.height || j0 < 0 || j0 >= input->Size.width)
                continue;
            sum += input->GetElement(i0, j0) * filter->GetElement(i, j);
        }
    }
    return sum;
}

__global__ void ConvDirectDist_global(CudaTensor* InputMatrix , CudaTensor* Filter , CudaTensor* OutputMatrix){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = d_ConvDirectDist(index_x, index_y, InputMatrix, Filter);

    __syncthreads();
      OutputMatrix->SetElement(index_x, index_y, sum);
    __syncthreads();


}

float* CudaConvDirectDist(float* InputMatrix , float* Filter , float* OutputMatrix, Tensorsize I, Tensorsize F,Tensorsize O){
     dim3 gridSize , blockSize;
    if(I.width < BLOCK_SIZE || I.height < BLOCK_SIZE ){
        gridSize = dim3(I.width, I.height, 1);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(I.width / BLOCK_SIZE, I.height / BLOCK_SIZE, 1);
        blockSize = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    } 
    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* CFilter = new CudaTensor(F.width, F.height , F.depth , Filter);
    CudaTensor* COutputMatrix = new CudaTensor(O.width, O.height , O.depth , OutputMatrix);
    

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ConvDirectDist_global<<< gridSize , blockSize>>>(CInputMatrix, CFilter, COutputMatrix);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    hipMemcpy(OutputMatrix, COutputMatrix->Data, O.width * O.height * O.depth * sizeof(double), hipMemcpyDeviceToHost);

    delete CInputMatrix;
    delete CFilter;
    delete COutputMatrix;

    return OutputMatrix;

}

//__device__ void d_ConvBackDist(CudaTensor& InputMatrix , CudaTensor& Filter , CudaTensor& OutputMatrix){}

//__global__ void ConvBackDist_global(CudaTensor& InputMatrix , CudaTensor& Filter , CudaTensor& OutputMatrix){}

//Tensor CudaConvBackDist(Tensor& InputMatrix , Tensor& Filter, Tensor& OutputMatrix){}