#include "hip/hip_runtime.h"
#include "CUDA/CudaModuleCalc.h"
#include "stdio.h"
#include <time.h> 
#include <iostream>

#define PAD 1
// calc conv module
#if __CUDA_ARCH__ < 600
__device__ double _atomicAdd(double* address, double val)
{
 unsigned long long int* address_as_ull =
 (unsigned long long int*)address;
 unsigned long long int old = *address_as_ull, assumed;

 do {
 assumed = old;
 old = atomicCAS(address_as_ull, assumed,
 __double_as_longlong(val +
 __longlong_as_double(assumed)));

 // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
 } while (assumed != old);

 return __longlong_as_double(old);
}
#endif

__device__ double d_ConvDirectDist(int index_x, int index_y ,int index_z , CudaTensor *input , CudaTensor* filter  ,CudaTensor* CBias){
    double sum = CBias->GetElement(0 , 0 , index_z);
    int Padding = PAD; //(filter->Size.width - 1) / 2;
    for (int i = 0; i < filter->Size.height; i++) {
        for (int j = 0; j < filter->Size.width; j++) {
            int i0 = index_y + i - Padding;
            int j0 = index_x + j - Padding;
            if (i0 < 0 || i0 >= input->Size.height || j0 < 0 || j0 >= input->Size.width)
                continue;
            for (int c = 0; c < input->Size.depth; c++)
                sum += input->GetElement(j0, i0 , c) * filter->Data[index_z * (filter->Size.height * filter->Size.width *input->Size.depth) + (filter->Size.width * input->Size.depth * i + j *input->Size.depth + c)];//GetElement(i, j , index_z + c);
        }
    }
    return sum;
}

__global__ void ConvDirectDist_global(CudaTensor* InputMatrix , CudaTensor* Filter , CudaTensor* OutputMatrix ,CudaTensor* CBias){

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;

    double sum = d_ConvDirectDist(index_x, index_y,index_z , InputMatrix, Filter ,CBias);

    __syncthreads();
      OutputMatrix->SetElement(index_x, index_y,index_z ,  sum);
    __syncthreads();


}

int CudaConvDirectDist(const std::vector<double> InputMatrix , std::vector<double> Filter , std::vector<double> &OutputMatrix,std::vector<double> Bias , Tensorsize I, Tensorsize F,Tensorsize O , int CountF){
    dim3 gridSize , blockSize;
   /* if(O.width < BLOCK_SIZE || O.height < BLOCK_SIZE ){

        gridSize = dim3(O.width, O.height, CountF);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(O.width / BLOCK_SIZE, O.height / BLOCK_SIZE, CountF);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
     //gridSize = dim3(O.width, O.height, CountF);
    // blockSize = dim3(1, 1, 1);
    gridSize = dim3(O.width, O.height , CountF);
    blockSize = dim3(1,1, 1);
    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* CFilter = new CudaTensor(F.width, F.height , F.depth * CountF  , Filter);
    CudaTensor* COutputMatrix = new CudaTensor(O.width, O.height , O.depth , OutputMatrix);
    CudaTensor* CBias = new CudaTensor( 1 , 1 , CountF , Bias);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ConvDirectDist_global<<< gridSize , blockSize>>>(CInputMatrix, CFilter, COutputMatrix , CBias);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(COutputMatrix->d_vec.begin(), COutputMatrix->d_vec.end(), OutputMatrix.begin());
    delete CInputMatrix;
    delete CFilter;
    delete COutputMatrix;
    delete CBias;
    return 1;

}

__global__ void  ConvCalcDeltaFilters_global(CudaTensor* InputMatrix , CudaTensor* d_Filers , CudaTensor* d_Bias, CudaTensor* deltas ,  Tensorsize Isize ,int CountF){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
        int Padding = PAD;
    	double delta = deltas->GetElement(index_x, index_y,index_z);
        for (int i = 0; i < d_Filers->Size.height; i++) {
            for (int j = 0; j < d_Filers->Size.width; j++) {
                int i0 = index_y + i - Padding;
                int j0 = index_x + j - Padding;
                if (i0 < 0 || i0 >= Isize.height || j0 < 0 || j0 >= Isize.width) continue;
                for (int c = 0; c < Isize.depth; c++){
                    int e = index_z * (d_Filers->Size.height * d_Filers->Size.width *Isize.depth) + (d_Filers->Size.width * Isize.depth * i + j *Isize.depth + c);
                    _atomicAdd(&(d_Filers->Data[e]) , (delta * InputMatrix->GetElement(j0, i0,c)));
                    
                }
            }
        }

        

        __syncthreads();
        _atomicAdd(&d_Bias->Data[index_z] ,  delta );
        __syncthreads();
}

 int CudaConvCalcDFilters(const std::vector<double> InputMatrix ,std::vector<double> &d_Filers , std::vector<double> &d_Bias ,std::vector<double> deltas , Tensorsize I, Tensorsize dF, Tensorsize _size , Tensorsize InputSize, int CountF){
    dim3 gridSize , blockSize;
  /*  if(_size.width < BLOCK_SIZE || _size.height < BLOCK_SIZE ){
        gridSize = dim3(_size.width, _size.height , CountF);
        blockSize = dim3(1,1, 1);
    }else{
        gridSize = dim3(_size.width / BLOCK_SIZE, _size.height / BLOCK_SIZE,  CountF);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(_size.width, _size.height , CountF);
    blockSize = dim3(1,1, 1);
    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* CdFilter = new CudaTensor(dF.width, dF.height , dF.depth * CountF  , d_Filers);
    CudaTensor* CdBias = new CudaTensor(1, 1 , CountF , d_Bias);
    CudaTensor* Cdeltas = new CudaTensor(_size.width, _size.height , _size.depth  , deltas);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    ConvCalcDeltaFilters_global<<< gridSize , blockSize>>>(CInputMatrix , CdFilter , CdBias ,Cdeltas ,InputSize , CountF);
    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(CdFilter->d_vec.begin(), CdFilter->d_vec.end(), d_Filers.begin());
    thrust::copy(CdBias->d_vec.begin(), CdBias->d_vec.end(), d_Bias.begin());
    delete CInputMatrix;
    delete CdFilter;
    delete CdBias;
    delete Cdeltas;
    return 1;

 }

__global__ void  ConvCalcDx_global(CudaTensor* Filter , CudaTensor* dX , CudaTensor* deltas ,Tensorsize _size ,int CountF){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
    int Padding = PAD;
	int pad = Filter->Size.width - 1 - Padding;
    //double sum = d_ConvDirectDist(index_x, index_y,index_z , InputMatrix, Filter ,CBias);
    double sum = 0;
		for (int i = 0; i < Filter->Size.width; i++) {
			for (int j = 0; j < Filter->Size.height; j++) {
				int i0 = index_y + i - pad; // height
				int j0 = index_x + j - pad; // width
				if (i0 < 0 || i0 >= _size.height || j0 < 0 || j0 >= _size.width)
					continue;
				for (int f = 0; f < CountF; f++){
                    //solved
                    int i_f =  Filter->Size.width - 1 - i;
                    int j_f =  Filter->Size.width - 1 - j;
                    int e = f * (Filter->Size.height * Filter->Size.width *dX->Size.depth) + (Filter->Size.width * dX->Size.depth * i_f + j_f *dX->Size.depth + index_z);
                    sum += Filter->Data[e] * deltas->GetElement(j0, i0, f); 
                }
			}
		}

    __syncthreads();
      dX->SetElement(index_x, index_y,index_z ,  sum);
    __syncthreads();


}

int CudaConvCalcDx(std::vector<double> Filter , std::vector<double> &dX,std::vector<double> deltas , Tensorsize F, Tensorsize del,Tensorsize Sdx ,Tensorsize _size, int CountF){

    dim3 gridSize , blockSize;
   /* if(Sdx.width < BLOCK_SIZE || Sdx.height < BLOCK_SIZE ){

        gridSize = dim3(Sdx.width, Sdx.height, CountF);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(Sdx.width / BLOCK_SIZE, Sdx.height / BLOCK_SIZE, CountF);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(Sdx.width, Sdx.height , Sdx.depth);
    blockSize = dim3(1,1, 1);
    CudaTensor* CdX = new CudaTensor(Sdx.width, Sdx.height , Sdx.depth , dX);
    CudaTensor* CFilter = new CudaTensor(F.width, F.height , F.depth * CountF  , Filter);
    CudaTensor* Cdeltas = new CudaTensor(_size.width, _size.height , _size.depth , deltas);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ConvCalcDx_global<<< gridSize , blockSize>>>(CFilter, CdX, Cdeltas ,_size,CountF);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(CdX->d_vec.begin(), CdX->d_vec.end(), dX.begin());
    delete CdX;
    delete CFilter;
    delete Cdeltas;
    return 1;
}
__global__ void  ActivationDirectDist_global(CudaTensor* InputMatrix , CudaTensor* OutputMatrix){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
    __syncthreads();
        if(InputMatrix->GetElement(index_x, index_y, index_z) > 0)
            OutputMatrix->SetElement(index_x, index_y, index_z , InputMatrix->GetElement(index_x, index_y, index_z));
        else OutputMatrix->SetElement(index_x, index_y, index_z , 0);
    __syncthreads();

}
int CudaActivationDirectDist(std::vector<double> InputMatrix , std::vector<double> &OutputMatrix , Tensorsize I ,Tensorsize O){
    dim3 gridSize , blockSize;
 /*   if(O.width < BLOCK_SIZE || O.height < BLOCK_SIZE ){
        gridSize = dim3(O.width, O.height, O.depth);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(O.width / BLOCK_SIZE, O.height / BLOCK_SIZE,  O.depth);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(O.width, O.height, O.depth);
    blockSize = dim3(1, 1, 1);
    //gridSize = dim3(O.width / BLOCK_SIZE, O.height / BLOCK_SIZE,  O.depth);
    //blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);

    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* COutputMatrix = new CudaTensor(O.width, O.height , O.depth , OutputMatrix);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ActivationDirectDist_global<<< gridSize , blockSize>>>(CInputMatrix, COutputMatrix);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(COutputMatrix->d_vec.begin(), COutputMatrix->d_vec.end(), OutputMatrix.begin());
    delete CInputMatrix;
    delete COutputMatrix;
    return 1;
}

__global__ void  ActivationBackDist_global(CudaTensor* InputMatrix , CudaTensor* DeltaMatrix , CudaTensor* OutputMatrix){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
    __syncthreads();
    //dX(i, j, k) = dout(i, j, k) * (X(i, j, k) > 0 ? 1 : 0); 
        if(InputMatrix->GetElement(index_x, index_y, index_z) > 0)
            OutputMatrix->SetElement(index_x, index_y, index_z , DeltaMatrix->GetElement(index_x, index_y, index_z));
        else OutputMatrix->SetElement(index_x, index_y, index_z , 0);
    __syncthreads();


}

int CudaActivationBackDist(std::vector<double> InputMatrix , std::vector<double> DeltaMatrix , std::vector<double> &OutputMatrix , Tensorsize I , Tensorsize D ,Tensorsize O){
    dim3 gridSize , blockSize;
  /*  if(O.width < BLOCK_SIZE || O.height < BLOCK_SIZE ){
        gridSize = dim3(O.width, O.height, O.depth);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(O.width / BLOCK_SIZE, O.height / BLOCK_SIZE,  O.depth);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(O.width, O.height, O.depth);
    blockSize = dim3(1, 1, 1);
    //gridSize = dim3(O.width / BLOCK_SIZE, O.height / BLOCK_SIZE,  O.depth);
    //blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* CDeltaMatrix = new CudaTensor(D.width, D.height , D.depth , DeltaMatrix);
    CudaTensor* COutputMatrix = new CudaTensor(O.width, O.height , O.depth , OutputMatrix);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    ActivationBackDist_global<<< gridSize , blockSize>>>(CInputMatrix,CDeltaMatrix, COutputMatrix);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(COutputMatrix->d_vec.begin(), COutputMatrix->d_vec.end(), OutputMatrix.begin());
    delete CInputMatrix;
    delete CDeltaMatrix;
    delete COutputMatrix;
    return 1;
}


__global__ void  PoolingDirectDist_global(CudaTensor* InputMatrix ,CudaTensor* Filter , CudaTensor* OutputMatrix, int scale){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
    __syncthreads();

    if(index_x % scale == 0 && index_y % scale == 0 ){
        int imax = index_y;
        int jmax = index_x; 
        double max = InputMatrix->GetElement(index_y, index_x, index_z);
        for (int y = index_y; y < index_y + scale; y++) {
            for (int x = index_x; x < index_x + scale; x++) {
                double value = InputMatrix->GetElement(y, x, index_z);
                
                Filter->SetElement(y, x, index_z , 0);
                if (value > max) {
                    max = value;
                    imax = y;
                    jmax = x;
                }
            }
        }
        __syncthreads();
        OutputMatrix->SetElement(index_y / scale, index_x / scale, index_z , max);
        Filter->SetElement(imax, jmax, index_z , 1);
    }
    __syncthreads();
}

int CudaPoolingDirectDist(const std::vector<double> InputMatrix , std::vector<double> &Filter , std::vector<double> &OutputMatrix, Tensorsize I, Tensorsize F,Tensorsize O , int scale){
    dim3 gridSize , blockSize;
   /* if(I.width < BLOCK_SIZE || I.height < BLOCK_SIZE ){
        gridSize = dim3(I.width, I.height, I.depth);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(I.width / BLOCK_SIZE, I.height / BLOCK_SIZE,  I.depth);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(I.width, I.height, I.depth);
    blockSize = dim3(1, 1, 1);
    //gridSize = dim3(I.width / BLOCK_SIZE, I.height / BLOCK_SIZE,  I.depth);
    //blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    CudaTensor* CInputMatrix = new CudaTensor(I.width, I.height , I.depth , InputMatrix);
    CudaTensor* CFilter = new CudaTensor(F.width, F.height , F.depth , Filter);
    CudaTensor* COutputMatrix = new CudaTensor(O.width, O.height , O.depth , OutputMatrix);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    PoolingDirectDist_global<<< gridSize , blockSize>>>(CInputMatrix, CFilter , COutputMatrix,scale);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(COutputMatrix->d_vec.begin(), COutputMatrix->d_vec.end(), OutputMatrix.begin());
    thrust::copy(CFilter->d_vec.begin(), CFilter->d_vec.end(), Filter.begin());
    delete CInputMatrix;
    delete CFilter;
    delete COutputMatrix;
    return 1;
}
/* + */
__global__ void  PoolingBackDist_global(CudaTensor* DeltaMatrix , CudaTensor* FilterMatrix , CudaTensor* dXMatrix, int scale){
    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;
    int index_z = blockIdx.z * blockDim.z + threadIdx.z;
    __syncthreads();
        dXMatrix->SetElement(index_y, index_x, index_z , DeltaMatrix->GetElement(index_y / scale, index_x / scale, index_z) * FilterMatrix->GetElement(index_y, index_x, index_z)); 
    __syncthreads();
}

int CudaPoolingBackDist(std::vector<double> deltas , std::vector<double> Filter , std::vector<double> &dX , Tensorsize del, Tensorsize F,Tensorsize Sdx, int scale){
    dim3 gridSize , blockSize;
   /* if(Sdx.width < BLOCK_SIZE || Sdx.height < BLOCK_SIZE ){
        gridSize = dim3(Sdx.width, Sdx.height, Sdx.depth);
        blockSize = dim3(1, 1, 1);
    }else{
        gridSize = dim3(Sdx.width / BLOCK_SIZE, Sdx.height / BLOCK_SIZE,  Sdx.depth);
        blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    }*/
    gridSize = dim3(Sdx.width, Sdx.height, Sdx.depth);
    blockSize = dim3(1, 1, 1);
    //gridSize = dim3(Sdx.width / BLOCK_SIZE, Sdx.height / BLOCK_SIZE,  Sdx.depth);
    //blockSize = dim3(BLOCK_SIZE,BLOCK_SIZE, 1);
    CudaTensor* CDeltaMatrix = new CudaTensor(del.width, del.height , del.depth , deltas);
    CudaTensor* CFilter = new CudaTensor(F.width, F.height , F.depth , Filter);
    CudaTensor* CdXMatrix = new CudaTensor(Sdx.width, Sdx.height , Sdx.depth , dX);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    PoolingBackDist_global<<< gridSize , blockSize>>>(CDeltaMatrix, CFilter , CdXMatrix,scale);

    hipEventRecord(stop);
    hipDeviceSynchronize(); 
    hipGetLastError();
    thrust::copy(CdXMatrix->d_vec.begin(), CdXMatrix->d_vec.end(), dX.begin());
    delete CDeltaMatrix;
    delete CFilter;
    delete CdXMatrix;
    return 1;
}


